#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addKernel(int *a, int *b, int *c, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        c[i] = a[i] + b[i];
}

extern "C" void launchKernel(int *a, int *b, int *c, int N) {
    int *d_a, *d_b, *d_c;
    size_t size = N * sizeof(int);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    addKernel<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}